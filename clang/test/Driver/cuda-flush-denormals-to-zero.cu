
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed
// -fcuda-flush-denormals-to-zero. This should be translated to
// -fdenormal-fp-math-f32=preserve-sign

// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_20 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fcuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=sm_70 -fno-cuda-flush-denormals-to-zero -nocudainc -nocudalib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test explicit argument, with CUDA offload kind
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fcuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fno-cuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test explicit argument, with HIP offload kind
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fcuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -fno-cuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fcuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -fno-cuda-flush-denormals-to-zero -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s

// Test the default changing with no argument based on the subtarget in HIP mode
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZ %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s


// Test multiple offload archs with different defaults.
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=MIXED-DEFAULT-MODE %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell -fcuda-flush-denormals-to-zero --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=FTZX2 %s
// RUN: %clang -x hip -no-canonical-prefixes -### -target x86_64-linux-gnu -c -march=haswell -fno-cuda-flush-denormals-to-zero --cuda-gpu-arch=gfx803 --cuda-gpu-arch=gfx900 -nocudainc -nogpulib %s 2>&1 | FileCheck -check-prefix=NOFTZ %s


// CPUFTZ-NOT: -fdenormal-fp-math

// FTZ-NOT: -fdenormal-fp-math-f32=
// FTZ: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"

// The default of ieee is omitted
// NOFTZ-NOT: "-fdenormal-fp-math"
// NOFTZ-NOT: "-fdenormal-fp-math-f32"

// MIXED-DEFAULT-MODE-NOT: -denormal-fp-math
// MIXED-DEFAULT-MODE: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// MIXED-DEFAULT-MODE-SAME: "-target-cpu" "gfx803"
// MIXED-DEFAULT-MODE-NOT: -denormal-fp-math

// FTZX2: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// FTZX2-SAME: "-target-cpu" "gfx803"
// FTZX2: "-fdenormal-fp-math-f32=preserve-sign,preserve-sign"
// FTZX2-SAME: "-target-cpu" "gfx900"
